#include "hip/hip_runtime.h"
#include "cuda_common.cuh"

__global__ void kern() {
	printf("Hello CUDA\n");
}

int main() {
	kern << <dim3(1), dim3(1) >> > ();
	gpuErrCheck(hipDeviceSynchronize());
	return 0;
}