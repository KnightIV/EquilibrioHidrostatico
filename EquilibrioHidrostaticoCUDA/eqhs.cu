#include "hip/hip_runtime.h"
#include "cuda_common.cuh"

#include <iostream>
#include <fstream>
#include <filesystem>

#include "eqhs_phys.cuh"

using std::cout;
using std::endl;
using std::string;
using std::ofstream;

struct SimProps {

	const double gridStep = 1'000;
	const double z_start = 0;
	const double z_end = 6e6;

	SimProps() {}
	SimProps(const double gridStep, const double z_start, const double z_end)
		: gridStep(gridStep), z_start(z_start), z_end(z_end) {
	}

	__host__ __device__ inline const long gridSize() const {
		return ((long)(z_end - z_start)) / (long)gridStep;
	}
};

struct FinalExportData {

	const double *altitudes, *temperature, *pressure, *density;
	const int size;

	FinalExportData(const double *altitudes, const double *temperature, const double *pressure, const double *density, const int size)
		: altitudes(altitudes), temperature(temperature), pressure(pressure), density(density), size(size) {
	}

	~FinalExportData() {
		delete[] altitudes;
		delete[] temperature;
		delete[] pressure;
		delete[] density;
	}
};

void exportDataCsv(const FinalExportData &exportData, const string outputFileName) {
	string resultsDir = std::filesystem::current_path().string() + "/results";
	if (!std::filesystem::is_directory(resultsDir) || !std::filesystem::exists(resultsDir)) {
		cout << "Creating results directory at " << resultsDir << "\n";
		std::filesystem::create_directory(resultsDir);
	}

	string outputFilePath = resultsDir + "/" + outputFileName;
	if (outputFilePath.find(".csv") == string::npos) {
		outputFilePath += ".csv";
	}

	cout << "Writing out results to " << outputFilePath << endl;

	ofstream outFile(outputFilePath);
	if (outFile.is_open()) {
		string header = "Altitude (z),Temperature (K),Pressure,Density\n";
#if 0
		cout << header;
#endif
		outFile << header;

		for (auto i = 0; i < exportData.size; i++) {
			double alt = exportData.altitudes[i];
			double temp = exportData.temperature[i];
			double pressure = exportData.pressure[i];
			double density = exportData.density[i];
#if 0
			cout << alt << ","
				<< temp << ","
				<< pressure << ","
				<< density << "\n";
#endif
			outFile << alt << ","
				<< temp << ","
				<< pressure << ","
				<< density << "\n";
		}
		outFile.close();
	} else {
		cout << "Unable to open file\n";
	}
}

__global__ void initAltitudeGrid(const SimProps *p, double *z) {
	const int gid = calc1Dgid();

	if (gid < p->gridSize()) {
		z[gid] = p->gridStep * gid;
	}
}

__global__ void integrate(const SimProps *p, const double *z, double *temperature, double *pressure, double *density) {
	const int gid = calc1Dgid();

	if (gid < p->gridSize()) {
		temperature[gid] = eqhs_phys::temperature(z[gid]);
		pressure[gid] = eqhs_phys::pressure(temperature[gid]);
		density[gid] = eqhs_phys::density(temperature[gid], pressure[gid]);
	}
}

int main() {
	SimProps props;
	const long sizeBytes = props.gridSize() * sizeof(double);

	SimProps *d_props;
	double *d_z;

	double *d_temperature, *d_pressure, *d_density;
	gpuErrCheck(hipMalloc((void **)&d_temperature, sizeBytes));
	gpuErrCheck(hipMalloc((void **)&d_pressure, sizeBytes));
	gpuErrCheck(hipMalloc((void **)&d_density, sizeBytes));

	gpuErrCheck(hipMalloc((void **)&d_props, sizeof(SimProps)));
	gpuErrCheck(hipMemcpy(d_props, &props, sizeof(SimProps), hipMemcpyHostToDevice));
	gpuErrCheck(hipMalloc((void **)&d_z, sizeBytes));

	dim3 block(WARP_SIZE * 16);
	dim3 grid((props.gridSize() / block.x) + 1);

	initAltitudeGrid << <grid, block >> > (d_props, d_z);
	gpuErrCheck(hipDeviceSynchronize());

	integrate << <grid, block >> > (d_props, d_z, d_temperature, d_pressure, d_density);
	gpuErrCheck(hipDeviceSynchronize());

	double *h_temperature = new double[props.gridSize()];
	double *h_pressure = new double[props.gridSize()];
	double *h_density = new double[props.gridSize()];
	double *h_z = new double[props.gridSize()];

	gpuErrCheck(hipMemcpy((void **)h_z, d_z, sizeBytes, hipMemcpyDeviceToHost));
	gpuErrCheck(hipMemcpy((void **)h_temperature, d_temperature, sizeBytes, hipMemcpyDeviceToHost));
	gpuErrCheck(hipMemcpy((void **)h_pressure, d_pressure, sizeBytes, hipMemcpyDeviceToHost));
	gpuErrCheck(hipMemcpy((void **)h_density, d_density, sizeBytes, hipMemcpyDeviceToHost));
	FinalExportData exportData(h_z, h_temperature, h_pressure, h_density, props.gridSize());
	exportDataCsv(exportData, "cudaResults.csv");

	gpuErrCheck(hipFree(d_props));
	gpuErrCheck(hipFree(d_z));
	gpuErrCheck(hipFree(d_temperature));
	gpuErrCheck(hipFree(d_pressure));
	gpuErrCheck(hipFree(d_density));
	return 0;
}