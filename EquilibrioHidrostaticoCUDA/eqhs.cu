#include "hip/hip_runtime.h"
#include "cuda_common.cuh"

#include <iostream>

#include "eqhs_phys.cuh"

using std::cout;
using std::endl;

struct SimProps {

	const double gridStep = 1'000;
	const double z_start = 0;
	const double z_end = 6e6;

	SimProps() {}
	SimProps(const double gridStep, const double z_start, const double z_end)
		: gridStep(gridStep), z_start(z_start), z_end(z_end) {
	}

	inline long gridSize() {
		return ((long)(z_end - z_start)) / (long)gridStep;
	}
};

__global__ void initAltitudeGrid(const SimProps *p, double *z) {
	int gid = (blockIdx.x * blockDim.x) + threadIdx.x;
	z[gid] = p->gridStep * gid;
}

__global__ void calcTempIntegral(const SimProps *p, const double *z, double *temperature, double *temperatureIntegral) {
	
}

int main() {
	SimProps props;
	const long sizeBytes = props.gridSize() * sizeof(double);

	SimProps *d_props;
	double *d_z;

	double *d_temperature, *d_temperatureIntegral, *d_pressure, *d_density;
	gpuErrCheck(hipMalloc((void **)&d_temperature, sizeBytes));
	gpuErrCheck(hipMalloc((void **)&d_temperatureIntegral, sizeBytes));
	gpuErrCheck(hipMalloc((void **)&d_pressure, sizeBytes));
	gpuErrCheck(hipMalloc((void **)&d_density, sizeBytes));

	gpuErrCheck(hipMalloc((void**) &d_props, sizeof(SimProps)));
	gpuErrCheck(hipMemcpy(d_props, &props, sizeof(SimProps), hipMemcpyHostToDevice));
	gpuErrCheck(hipMalloc((void**) &d_z, sizeBytes));
	
	dim3 block(WARP_SIZE * 16);
	dim3 grid((props.gridSize() / block.x) + 1);
	//dim3 block(1), grid(1);
	initAltitudeGrid << <grid, block >> > (d_props, d_z);
	gpuErrCheck(hipDeviceSynchronize());

	calcTempIntegral << <grid, block >> > (d_props, d_z, d_temperature, d_temperatureIntegral);
	gpuErrCheck(hipDeviceSynchronize());

	gpuErrCheck(hipFree(d_props));
	gpuErrCheck(hipFree(d_z));
	gpuErrCheck(hipFree(d_temperature));
	gpuErrCheck(hipFree(d_temperatureIntegral));
	gpuErrCheck(hipFree(d_pressure));
	gpuErrCheck(hipFree(d_density));
	return 0;
}